#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "wav.hpp"
#include "stopwatch.hpp"
#include "manip.hpp"
#include "utils.hpp"

#include <stdio.h>
#include <vector>
#include <fstream>
#include <string>
#include <thread>

// --------------
std::ofstream logfile("results.log");
const int numCudaCores = 4864; // RTX 3060 Ti (4864 CUDA Cores)
std::string outputFilename = "";
int numSegments = 4;
// --------------

__global__ void halveCudaHelper(short* data) {
    data[blockIdx.x * blockDim.x + threadIdx.x] /= 2;
}

void processCuda(std::vector<short>& samples, int threadsPerBlock) {
    short* data = samples.data();
    size_t numSamples = samples.size();

    hipMalloc(&data, numSamples * sizeof(short));
    hipMemcpy(data, samples.data(), numSamples, hipMemcpyHostToDevice);

    int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
    halveCudaHelper<<<blocksPerGrid, threadsPerBlock>>> (data);

    hipMemcpy(samples.data(), data, numSamples, hipMemcpyDeviceToHost);
    hipFree(data);
}


void processCudaSegmented(std::vector<short>& samples, int numSegments) {
    int segmentSize = samples.size() / numSegments;
    std::vector<std::thread> threads;

    for (int i = 0; i < numSegments; i++)
    {
        int start = i * segmentSize;
        int end = (i + 1) * segmentSize;
        threads.push_back(std::thread([start, end, numSegments, &samples]()
            {
                std::vector<short> segment(samples.begin() + start, samples.begin() + end);
                processCuda(segment, numCudaCores / numSegments);
            }
        ));
    }

    for (auto& t : threads)
        t.join();
}

void benchmark(const std::string& filename) {
    stopwatch sw;
    WAV w(filename);
    w.init();
    auto original = std::vector<short>(w.data);

    printf("Processing %s, size: %.2f MB.\n", filename.c_str(), filesize(filename));

    /*

    // 1. CUDA
    printf("Started: CUDA\n");
    sw.start();
    processCuda(w.data, numCudaCores);
    sw.stop();

    printf("CUDA: %d [ms].\n", sw.elapsed());
    log("CUDA: " + filename + " " + std::to_string(sw.elapsed()) + " [ms]", logfile);
    sw.reset();
    w.save("results/cuda_" + filename);

    */

    if (numSegments > 1) {
        if (numSegments > w.data.size())
        {
            fprintf(stderr, "Segmenting failed! num_segments > num_samples (%d > %d)\n", numSegments, w.data.size());
            exit(1);
        }

        // 2. CUDA + Segmenting
        printf("Started: CUDA + Segmenting\n");
        sw.start();
        processCudaSegmented(w.data, numSegments);
        sw.stop();
        printf("CUDA + Segmenting (%d segments): %d [ms].\n", numSegments, sw.elapsed());
        log("CUDA + Segmenting: " + filename + " " + std::to_string(sw.elapsed()) + " [ms]", logfile);
        sw.reset();
        w.save("results/cudasegmenting_" + filename);
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        logfile.close();
        exit(1);
    }
}

void testall() {
    std::vector<std::string> filenames = { "long.wav", "medium.wav", "short.wav", "vshort.wav" };
    for (auto& filename : filenames)
        for (int i = 0; i < 4; i++) {
            benchmark(filename);
        }
}

int main(int argC, char **argV)
{
    //if (argC != 3 && argC != 4)
    //{
    //    std::cout << "Usage: "
    //        << "<path to .wav file> "
    //        << "<output file> "
    //        << "<optional: # of segments> "
    //        << std::endl;
    //    return 0;
    //}
    //else if (argC == 4)
    //    numSegments = std::atoi(argV[3]);

    /*for (int i = 0; i < 4; i++) {
        benchmark(std::string(argV[1]));
    }*/

    testall();

    logfile.close();
    hipDeviceReset();
    
    std::cout << "Press any key to exit..." << std::endl;
    getchar();
    return 0;
}